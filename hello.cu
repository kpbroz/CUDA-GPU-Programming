
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * The addition of `__global__` signifies that this function
 * should be launced on the GPU.
 */

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{

  /* Task 1: print the messages from functins helloCPU and helloGPU */

  printf("Task 1:\n");
  helloCPU();


  /*
   * Add an execution configuration with the <<<...>>> syntax
   * will launch this function as a kernel on the GPU.
   */

  helloGPU<<<1, 1>>>();

  /*
   * `cudaDeviceSynchronize` will block the CPU stream until
   * all GPU kernels have completed.
   */

  hipDeviceSynchronize();

  printf("\n");

  /* Task 2: Hello from the GPU prints before Hello from the CPU */

  printf("Task 2:\n");
  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize();
  helloCPU();

  printf("\n");

  /* Task 3: Hello from the GPU prints twice, once before Hello from the CPU, and once after. */

  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize();
  helloCPU();
  helloGPU<<<1, 1>>>();
}
