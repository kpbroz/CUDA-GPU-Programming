
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{

    /* Task 1: Refactor the firstParallel function to launch as a CUDA kernel on the GPU */
    printf("Task 1:\n");
    firstParallel<<<1,1>>>();
    hipDeviceSynchronize();


    /* Task 2: Refactor the firstParallel kernel to execute in parallel on 5 threads, all executing in a single thread block. */
    printf("Task 2:\n");
    firstParallel<<<1,5>>>();
    hipDeviceSynchronize();

    /* Task 3: Refactor the firstParallel kernel again, this time to execute in parallel inside 5 thread blocks, each containing 5 threads. */
    printf("Task 3:\n");
    firstParallel<<<5,5>>>();
    hipDeviceSynchronize();

}
